#include "hip/hip_runtime.h"

//General libraries
#include <time.h>
#include <iostream>
#include <math.h>
#include <fstream>
#include <stdio.h>
#include <string.h>
#include <sstream>

//Custom headers
#include "Phase_point.h"
#include "Lagrangian_points.h"
#include "global_var.hpp"
#include "er3bp_system.hpp"

//GSL libraries
#include <gsl/gsl_matrix.h>
#include <gsl/gsl_eigen.h>
#include <gsl/gsl_complex.h>
#include <gsl/gsl_complex_math.h>
#include <gsl/gsl_cblas.h>
#include <gsl/gsl_blas.h>


//Boost libraries
#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>
#include <boost/program_options.hpp>

//Thrust libraries
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/remove.h>


using namespace boost::numeric::odeint;

typedef double value_type;
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;
//typedef runge_kutta_fehlberg78< state_type , value_type , state_type , value_type , thrust_algebra , thrust_operations > stepper_type;
typedef runge_kutta_dopri5< state_type , value_type , state_type , value_type , thrust_algebra , thrust_operations > stepper_type;
//typedef bulirsch_stoer< state_type , value_type , state_type , value_type , thrust_algebra , thrust_operations > stepper_type; //FIXME
//typedef runge_kutta_cash_karp54< state_type , value_type , state_type , value_type , thrust_algebra , thrust_operations > stepper_type;


//typedef runge_kutta_dopri5< value_type , state_type , value_type , state_type , thrust_algebra , thrust_operations > stepper;
typedef controlled_runge_kutta< stepper_type , custom_error_checker > controlled_stepper ;
//typedef dense_output_runge_kutta< controlled_stepper > dense_output_stepper;


typedef thrust::device_vector< int> int_vec_type;
    // Integrator precision
double abs_tol=1e-6;
double rel_tol=1e-6;

typedef std::vector<double> double1d;
typedef std::vector<double1d> double2d;
typedef std::vector<double2d> double3d;
typedef std::vector<double3d> double4d;

typedef std::vector<phase_point> pp1d;
typedef std::vector<pp1d> pp2d;
typedef std::vector<pp2d> pp3d;
typedef std::vector<pp3d> pp4d;

/* Functions declarations */
struct Lagrangian_points* Lagr(double mu);
double Omega (double x, double y, double mu);
int id_calc(int *arr);
int create_ic_vector (double c_max, double c_min, int nc, std::vector<double>& c0);
int create_ic_vector (double C0, double dc, std::vector<double>& c0);
int create_missing_vector(int id, double1d& x_0, double1d& y_0, double1d& vx_0, double1d& vy_0, double1d& e_0, double4d& filter_integration, double Ki );
int create_integration_vector (thrust::device_vector< double >& X_state, int dim, double1d x_0, double1d y_0, double1d vx_0, double1d vy_0, double1d e_0, double4d& filter_integration, double4d& filter_ftle);

int RUN (double t0)
{
double tempo;

  // Computation parameters
double Tf=t0+DT;
double Ki;
Ki=1+ecc*cos(t0);

int id=id_calc(flags);

double4d ftle(nx, double3d(ny, double2d(n1, double1d(n2,0) ) ));
double4d filter_integration(nx, double3d(ny, double2d(n1, double1d(n2,0) ) ));
double4d filter_ftle(nx, double3d(ny, double2d(n1, double1d(n2,0) ) ));
double4d filter_collisions(nx, double3d(ny, double2d(n1, double1d(n2,0) ) ));

pp4d pf(nx, pp3d(ny, pp2d(n1, pp1d(n2) ) ));

printf("ftle dim=%i, %i, %i, %i\n",ftle.size(),ftle[1].size(), ftle[1][1].size(), ftle[1][1][1].size());
//printf("id=%i\n",id);

double1d x_0(nx);
double1d y_0(ny);
double1d vx_0(nvx);
double1d vy_0(nvy);
double1d e_0(ne);
char file_name[100]=""; //String that will become the file name
char s_temp[50]="";
sprintf(file_name,"ftle_ell_mu=%.4f_ecc=%.2f",mu,ecc);
char file_header[200]=""; //String that will be printed in the file
sprintf(file_header,"mu=%.4f\necc=%.4f\nDT=%.2f\nt0=%.2f\nn_frames=%i\nid=%i\n",mu,ecc,DT,t0,n_frames,id);

//Create vectors of initial conditions

    //Create x_0
/*
if x is a visualization variable the initial condition vector can be computed from x_max, x_min and nx.
If it is a fixed variable, determine dx by comparison and compute initial condition vector using dx and the middle value X_0.
*/
if (flags[0]==1)
{
    create_ic_vector(x_max, x_min, nx, x_0);
    sprintf(s_temp,"nx=%i",nx);
    strcat(file_name,s_temp);
    strcat(file_header,s_temp);
    strcat(file_header,"\n");
    //printf("dx=%.4f\n",dx);
}
else
{
    if (flags[0]==2)
    {
        sprintf(s_temp,"x0=%.2f",X_0);
        strcat(file_name,s_temp);
        sprintf(s_temp,"x_0=%.6f",X_0);
        strcat(file_header,s_temp);
        strcat(file_header,"\n");
        if (flags[1]==1){dx=dy;}
        else{   if (flags[2]==1){dx=dvx;} //FIXME dimensioni diverse!
                else{   if (flags[3]==1){dx=dvy;} //FIXME dimensioni diverse!!
                        else {dx=de;}   //FIXME dimensioni diverse!!
                    }
            }
        create_ic_vector(X_0, dx, x_0);
    }
}

//Create y_0
if (flags[1]==1)
{
    create_ic_vector(y_max, y_min, ny, y_0);
    sprintf(s_temp,"ny=%i",ny);
    strcat(file_name,s_temp);
    strcat(file_header,s_temp);
    strcat(file_header,"\n");
}
else
{
    if (flags[1]==2)
    {
        sprintf(s_temp,"y0=%.2f",Y_0);
        strcat(file_name,s_temp);
        sprintf(s_temp,"y_0=%.6f",Y_0);
        strcat(file_header,s_temp);
        strcat(file_header,"\n");
        if (flags[0]==1){dy=dx;}
        else{   if (flags[2]==1){dy=dvx;} //FIXME dimensioni diverse!
                else{   if (flags[3]==1){dy=dvy;} //FIXME dimensioni diverse!!
                        else {dy=de;}   //FIXME dimensioni diverse!!
                    }
            }
        create_ic_vector(Y_0, dy, y_0);
    }
}

//Create vx_0
if (flags[2]==1)
{
    create_ic_vector(vx_max, vx_min, nvx, vx_0);
    sprintf(s_temp,"nvx=%i",nvx);
    strcat(file_name,s_temp);
    strcat(file_header,s_temp);
    strcat(file_header,"\n");
}
else
{
    if (flags[2]==2)
    {
        sprintf(s_temp,"vx0=%.2f",VX_0);
        strcat(file_name,s_temp);
        sprintf(s_temp,"vx_0=%.6f",VX_0);
        strcat(file_header,s_temp);
        strcat(file_header,"\n");
        if (flags[0]==1){dvx=dx;}   //FIXME dimensioni diverse!
        else{   if (flags[1]==1){dvx=dy;} //FIXME dimensioni diverse!
                else{   if (flags[3]==1){dvx=dvy;}
                        else {dvx=de;}   //FIXME dimensioni diverse!!
                    }
            }
        create_ic_vector(VX_0, dvx, vx_0);
    }
}

//Create vy_0
if (flags[3]==1)
{
    create_ic_vector(vy_max, vy_min, nvy, vy_0);
    sprintf(s_temp,"nvy=%i",nvy);
    strcat(file_name,s_temp);
    strcat(file_header,s_temp);
    strcat(file_header,"\n");
}
else
{
    if (flags[3]==2)
    {
        sprintf(s_temp,"vy0=%.2f",VY_0);
        strcat(file_name,s_temp);
        sprintf(s_temp,"vy_0=%.6f",VY_0);
        strcat(file_header,s_temp);
        strcat(file_header,"\n");
        if (flags[0]==1){dvy=dx;}   //FIXME dimensioni diverse!!
        else{   if (flags[1]==1){dvy=dy;} //FIXME dimensioni diverse!
                else{   if (flags[2]==1){dvy=dvx;}
                        else {dvy=de;}   //FIXME dimensioni diverse!!
                    }
            }
        create_ic_vector(VY_0, dvy, vy_0);
    }
}

//Create e_0
if (flags[4]==1)
{
    create_ic_vector(e_max, e_min, ne, e_0);
    sprintf(s_temp,"ne=%i",ne);
    strcat(file_name,s_temp);
    strcat(file_header,s_temp);
    strcat(file_header,"\n");
}
else
{
    if (flags[4]==2)
    {
        sprintf(s_temp,"e0=%.2f",E_0);
        strcat(file_name,s_temp);
        sprintf(s_temp,"e_0=%.6f",E_0);
        strcat(file_header,s_temp);
        strcat(file_header,"\n");
        if (flags[0]==1){de=dx;} //FIXME dimensioni diverse!!
        else{   if (flags[1]==1){de=dy;} //FIXME dimensioni diverse!
                else{   if (flags[2]==1){de=dvx;} //FIXME dimensioni diverse!!
                        else {de=dvy;}   //FIXME dimensioni diverse!!
                    }
            }
        create_ic_vector(E_0, de, e_0);
    }
}

// Create missing vector
int c=0;
if (flags[0]==0){std::cout<<"Please provide a value for both x and y"; return 1;} /* x and y must always be provided as gridded or fixed variables*/
if (flags[1]==0){std::cout<<"Please provide a value for both x and y"; return 1;} /* x and y must always be provided as gridded or fixed variables*/

int dim;
dim=create_missing_vector(id, x_0, y_0, vx_0, vy_0, e_0, filter_integration, Ki);

    //Construct vector for integration
double perc=((double)dim/n_tot)*100;
printf("Points to be integrated=%i\t %.1f %% n_tot\n",dim,perc);

thrust::device_vector< double > X_state(5*dim,0); //5: x,y,vx,vy,e
create_integration_vector (X_state, dim, x_0, y_0, vx_0, vy_0, e_0, filter_integration, filter_ftle);

int_vec_type Collisions(dim,0);
er3bp_system sistema (dim, Collisions);
clock_t start=clock();
double steps;

//steps=integrate_adaptive( make_controlled(abs_tol , rel_tol, stepper_type() ), sistema , X_state , t0 , Tf , 1e-6);
//dense_output_stepper s(controlled_stepper(custom_controller( Collisions )));
//steps=integrate_adaptive( dense_output_stepper, sistema , X_state , t0 , Tf , 1e-6);
controlled_stepper s(Collisions);
steps=integrate_adaptive( s, sistema , X_state , t0 , Tf , 1e-6);

//std::cout<<steps<<'\n';
clock_t end_t=clock();
tempo=((double)(end_t-start))/CLOCKS_PER_SEC;
std::cout<<"integration time "<<tempo<<'\n';

    //Deconstruct integration results
start=clock();
c=0;
for (int i=0; i<nx; i++)
    {
        for (int j=0; j<ny; j++)
        {
            for (int k=0; k<n1; k++)
            {
                for (int l=0; l<n2; l++)
                {
                    pf[i][j][k][l].x=0;
                    pf[i][j][k][l].y=0;
                    pf[i][j][k][l].vx=0;
                    pf[i][j][k][l].vy=0;
                    pf[i][j][k][l].e=0;
                    if (filter_integration[i][j][k][l])
                    {
                        if (Collisions[c]==0)
                        {
                            pf[i][j][k][l].x=X_state[c];
                            pf[i][j][k][l].y=X_state[c+dim];
                            pf[i][j][k][l].vx=X_state[c+2*dim];
                            pf[i][j][k][l].vy=X_state[c+3*dim];
                            pf[i][j][k][l].e=0.5*((X_state[c+2*dim]*X_state[c+2*dim])+
                                                  (X_state[c+3*dim]*X_state[c+3*dim]))-
                                                  Omega(X_state[c],X_state[c+dim],mu)/(1+ecc*cos(Tf));
                        }
                        filter_collisions[i][j][k][l]=Collisions[c];
                        c=c+1;
                    }
                    /*
                    Do not compute ftle on the border of the phase space
                    */
                    if (i==0 || i==nx-1 || j==0 || j==ny-1 || k==0 || k==n1-1 || l==0 || l==n2-1)
                    {
                        filter_ftle[i][j][k][l]=0;
                    }
                    /*
                    Do not compute ftle for points in the neighbourhood of a not integrated point
                    */
                    else
                    {
                        if (filter_integration[i][j][k][l]==0 || filter_collisions[i][j][k][l]==1)
                        {
                        filter_ftle[i][j][k][l]=0;
                        filter_ftle[i+1][j][k][l]=0;
                        filter_ftle[i-1][j][k][l]=0;
                        filter_ftle[i][j+1][k][l]=0;
                        filter_ftle[i][j-1][k][l]=0;
                        filter_ftle[i][j][k+1][l]=0;
                        filter_ftle[i][j][k-1][l]=0;
                        filter_ftle[i][j][k][l+1]=0;
                        filter_ftle[i][j][k][l-1]=0;
                        }
                    }
                }
            }
        }
    }

end_t=clock();
tempo=((double)(end_t-start))/CLOCKS_PER_SEC;
std::cout<<"Deconstruction time "<<tempo<<'\n';

    //FTLE computation

gsl_matrix *dphi=gsl_matrix_alloc (4,4);
gsl_matrix *C=gsl_matrix_alloc (4,4);
double l_max;
gsl_vector *eval = gsl_vector_alloc (4);
gsl_eigen_symm_workspace * w = gsl_eigen_symm_alloc (4);

for (int i=0; i<nx; i++)
    {
        for (int j=0; j<ny; j++)
        {
            for (int k=0; k<n1; k++)
            {
                for (int l=0; l<n2; l++)
                {
                    //ftle[i][j][k][l]=0;
                    //std::cout<<i<<'\t'<<j<<'\t'<<k<<'\t'<<l<<'\n';
                    if (filter_ftle[i][j][k][l])
                    {
                        gsl_matrix_set(dphi,0,0,(pf[i+1][j][k][l].x-pf[i-1][j][k][l].x)/(2*dx));
                        gsl_matrix_set(dphi,0,1,(pf[i][j+1][k][l].x-pf[i][j-1][k][l].x)/(2*dy));
                        gsl_matrix_set(dphi,1,0,(pf[i+1][j][k][l].y-pf[i-1][j][k][l].y)/(2*dx));
                        gsl_matrix_set(dphi,1,1,(pf[i][j+1][k][l].y-pf[i][j-1][k][l].y)/(2*dy));

                        if (flags[2]==0) //k-->vy   l-->e
                        {
                            gsl_matrix_set(dphi,0,2,(pf[i][j][k+1][l].x-pf[i][j][k-1][l].x)/(2*dvy));
                            gsl_matrix_set(dphi,0,3,(pf[i][j][k][l+1].x-pf[i][j][k][l-1].x)/(2*de));

                            gsl_matrix_set(dphi,1,2,(pf[i][j][k+1][l].y-pf[i][j][k-1][l].y)/(2*dvy));
                            gsl_matrix_set(dphi,1,3,(pf[i][j][k][l+1].y-pf[i][j][k][l-1].y)/(2*de));

                            gsl_matrix_set(dphi,2,0,(pf[i+1][j][k][l].vy-pf[i-1][j][k][l].vy)/(2*dx));
                            gsl_matrix_set(dphi,2,1,(pf[i][j+1][k][l].vy-pf[i][j-1][k][l].vy)/(2*dy));
                            gsl_matrix_set(dphi,2,2,(pf[i][j][k+1][l].vy-pf[i][j][k-1][l].vy)/(2*dvy));
                            gsl_matrix_set(dphi,2,3,(pf[i][j][k][l+1].vy-pf[i][j][k][l-1].vy)/(2*de));

                            gsl_matrix_set(dphi,3,0,(pf[i+1][j][k][l].e-pf[i-1][j][k][l].e)/(2*dx));
                            gsl_matrix_set(dphi,3,1,(pf[i][j+1][k][l].e-pf[i][j-1][k][l].e)/(2*dy));
                            gsl_matrix_set(dphi,3,2,(pf[i][j][k+1][l].e-pf[i][j][k-1][l].e)/(2*dvy));
                            gsl_matrix_set(dphi,3,3,(pf[i][j][k][l+1].e-pf[i][j][k][l-1].e)/(2*de));
                        }

                        if (flags[3]==0)
                        {
                            gsl_matrix_set(dphi,0,2,(pf[i][j][k+1][l].x-pf[i][j][k-1][l].x)/(2*dvx));
                            gsl_matrix_set(dphi,0,3,(pf[i][j][k][l+1].x-pf[i][j][k][l-1].x)/(2*de));

                            gsl_matrix_set(dphi,1,2,(pf[i][j][k+1][l].y-pf[i][j][k-1][l].y)/(2*dvx));
                            gsl_matrix_set(dphi,1,3,(pf[i][j][k][l+1].y-pf[i][j][k][l-1].y)/(2*de));

                            gsl_matrix_set(dphi,2,0,(pf[i+1][j][k][l].vx-pf[i-1][j][k][l].vx)/(2*dx));
                            gsl_matrix_set(dphi,2,1,(pf[i][j+1][k][l].vx-pf[i][j-1][k][l].vx)/(2*dy));
                            gsl_matrix_set(dphi,2,2,(pf[i][j][k+1][l].vx-pf[i][j][k-1][l].vx)/(2*dvx));
                            gsl_matrix_set(dphi,2,3,(pf[i][j][k][l+1].vx-pf[i][j][k][l-1].vx)/(2*de));

                            gsl_matrix_set(dphi,3,0,(pf[i+1][j][k][l].e-pf[i-1][j][k][l].e)/(2*dx));
                            gsl_matrix_set(dphi,3,1,(pf[i][j+1][k][l].e-pf[i][j-1][k][l].e)/(2*dy));
                            gsl_matrix_set(dphi,3,2,(pf[i][j][k+1][l].e-pf[i][j][k-1][l].e)/(2*dvx));
                            gsl_matrix_set(dphi,3,3,(pf[i][j][k][l+1].e-pf[i][j][k][l-1].e)/(2*de));
                        }

                        if (flags[4]==0)
                        {
                            gsl_matrix_set(dphi,0,2,(pf[i][j][k+1][l].x-pf[i][j][k-1][l].x)/(2*dvx));
                            gsl_matrix_set(dphi,0,3,(pf[i][j][k][l+1].x-pf[i][j][k][l-1].x)/(2*dvy));

                            gsl_matrix_set(dphi,1,2,(pf[i][j][k+1][l].y-pf[i][j][k-1][l].y)/(2*dvx));
                            gsl_matrix_set(dphi,1,3,(pf[i][j][k][l+1].y-pf[i][j][k][l-1].y)/(2*dvy));

                            gsl_matrix_set(dphi,2,0,(pf[i+1][j][k][l].vx-pf[i-1][j][k][l].vx)/(2*dx));
                            gsl_matrix_set(dphi,2,1,(pf[i][j+1][k][l].vx-pf[i][j-1][k][l].vx)/(2*dy));
                            gsl_matrix_set(dphi,2,2,(pf[i][j][k+1][l].vx-pf[i][j][k-1][l].vx)/(2*dvx));
                            gsl_matrix_set(dphi,2,3,(pf[i][j][k][l+1].vx-pf[i][j][k][l-1].vx)/(2*dvy));

                            gsl_matrix_set(dphi,3,0,(pf[i+1][j][k][l].vy-pf[i-1][j][k][l].vy)/(2*dx));
                            gsl_matrix_set(dphi,3,1,(pf[i][j+1][k][l].vy-pf[i][j-1][k][l].vy)/(2*dy));
                            gsl_matrix_set(dphi,3,2,(pf[i][j][k+1][l].vy-pf[i][j][k-1][l].vy)/(2*dvx));
                            gsl_matrix_set(dphi,3,3,(pf[i][j][k][l+1].vy-pf[i][j][k][l-1].vy)/(2*dvy));
                        }

                        /* gsl_blas_dgemm(CblasTrans/CblasNoTrans,CblasTrans/CblasNoTrans,alpha,A,B,beta,C)*/ // alpha*A*B+beta*C
                        gsl_blas_dgemm(CblasTrans,CblasNoTrans,1.0,dphi,dphi,0,C);

                        gsl_eigen_symm (C, eval, w);

                        gsl_vector_set(eval,0,fabs(gsl_vector_get(eval,0)));
                        gsl_vector_set(eval,1,fabs(gsl_vector_get(eval,1)));
                        gsl_vector_set(eval,2,fabs(gsl_vector_get(eval,2)));
                        gsl_vector_set(eval,3,fabs(gsl_vector_get(eval,3)));
                        l_max=gsl_vector_max(eval);
                        ftle[i][j][k][l]=1/fabs(DT)*log(sqrt(l_max));
                    }
                }
            }
        }
    }

/*
Output file structure:

coordinate1   coordinate2   (coordinate3)  (coordinate4)    ftle

Number of coordinates varies with the number of gridded variables. Coordinates always appear in this order: x, y, vx, vy, e.
*/
FILE *ftle_stream;
sprintf(s_temp,"t0=%.2f",t0);
strcat(file_name,s_temp);
strcat(file_name,".txt");
sprintf(s_temp,"### Beginning of FTLE data ###\n");
strcat(file_header,s_temp);
ftle_stream=fopen(file_name,"w");
fprintf(ftle_stream,file_header);

if (id==11220 || id==11202 || id==11022)
{
    for (int i=0; i<nx; i++)
    {
        for (int j=0; j<ny; j++)
        {
            fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\n", x_0[i], y_0[j], ftle[i][j][1][1]);
        }
    }
}

if(id==12120 || id==12102)
{
    for (int i=0; i<nx; i++)
    {
        for (int j=0; j<nvx; j++)
        {
            fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\n", x_0[i], vx_0[j], ftle[i][1][j][1]);
        }
    }
}

if(id==12210)
{
    for (int i=0; i<nx; i++)
        {
            for (int j=0; j<nvy; j++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\n", x_0[i], vy_0[j], ftle[i][1][1][j]);
            }
        }
}

if(id==12012)
{
    for (int i=0; i<nx; i++)
        {
            for (int j=0; j<nvy; j++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\n", x_0[i], vy_0[j], ftle[i][1][j][1]);
            }
        }
}

if(id==12201 || id==12021)
{
    for (int i=0; i<nx; i++)
        {
            for (int j=0; j<ne; j++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\n", x_0[i], e_0[j], ftle[i][1][1][j]);
            }
        }
}

if (id==21120 || id==21102)
{
    for (int i=0; i<ny; i++)
    {
        for (int j=0; j<nvx; j++)
        {
            fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\n", y_0[i], vx_0[j], ftle[1][i][j][1]);
        }
    }
}

if (id==21012)
{
    for (int i=0; i<ny; i++)
    {
        for (int j=0; j<nvy; j++)
        {
            fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\n", y_0[i], vy_0[j], ftle[1][i][j][1]);
        }
    }
}

if (id==21210)
{
    for (int i=0; i<ny; i++)
    {
        for (int j=0; j<nvy; j++)
        {
            fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\n", y_0[i], vy_0[j], ftle[1][i][1][j]);
        }
    }
}

if (id==21201 || id==21021)
{
    for (int i=0; i<ny; i++)
    {
        for (int j=0; j<ne; j++)
        {
            fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\n", y_0[i], e_0[j], ftle[1][i][1][j]);
        }
    }
}

if (id==22110)
{
    for (int i=0; i<nvx; i++)
    {
        for (int j=0; j<nvy; j++)
        {
            fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\n", vx_0[i], vy_0[j], ftle[1][1][i][j]);
        }
    }
}

if (id==22101)
{
    for (int i=0; i<nvx; i++)
    {
        for (int j=0; j<ne; j++)
        {
            fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\n", vx_0[i], e_0[j], ftle[1][1][i][j]);
        }
    }
}

if (id==22011)
{
    for (int i=0; i<nvy; i++)
    {
        for (int j=0; j<ne; j++)
        {
            fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\n", vx_0[i], e_0[j], ftle[1][1][i][j]);
        }
    }
}
    //3D Cases
if (id==11120 || id==11102)
{
    for (int i=0; i<nx; i++)
    {
        for (int j=0; j<ny; j++)
        {
            for (int k=0; k<nvx; k++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\n", x_0[i], y_0[j], vx_0[k], ftle[i][j][k][1]);
            }
        }
    }
}

if (id==11210)
{
    for (int i=0; i<nx; i++)
    {
        for (int j=0; j<ny; j++)
        {
            for (int k=0; k<nvy; k++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\n", x_0[i], y_0[j], vy_0[k], ftle[i][j][1][k]);
            }
        }
    }
}

if (id==11012)
{
    for (int i=0; i<nx; i++)
    {
        for (int j=0; j<ny; j++)
        {
            for (int k=0; k<nvy; k++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\n", x_0[i], y_0[j], vy_0[k], ftle[i][j][k][1]);
            }
        }
    }
}

if (id==11021 || id==11201)
{
    for (int i=0; i<nx; i++)
    {
        for (int j=0; j<ny; j++)
        {
            for (int k=0; k<ne; k++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\n", x_0[i], y_0[j], e_0[k], ftle[i][j][1][k]);
            }
        }
    }
}

if (id==12110)
{
    for (int i=0; i<nx; i++)
    {
        for (int j=0; j<nvx; j++)
        {
            for (int k=0; k<nvy; k++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\n", x_0[i], vx_0[j], vy_0[k], ftle[i][1][j][k]);
            }
        }
    }
}

if (id==12101)
{
    for (int i=0; i<nx; i++)
    {
        for (int j=0; j<nvx; j++)
        {
            for (int k=0; k<ne; k++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\n", x_0[i], vx_0[j], e_0[k], ftle[i][1][j][k]);
            }
        }
    }
}

if (id==12011)
{
    for (int i=0; i<nx; i++)
    {
        for (int j=0; j<nvy; j++)
        {
            for (int k=0; k<ne; k++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\n", x_0[i], vy_0[j], e_0[k], ftle[i][1][j][k]);
            }
        }
    }
}

if (id==12011)
{
    for (int i=0; i<nx; i++)
    {
        for (int j=0; j<nvy; j++)
        {
            for (int k=0; k<ne; k++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\n", x_0[i], vy_0[j], e_0[k], ftle[i][1][j][k]);
            }
        }
    }
}

if (id==21110)
{
    for (int i=0; i<ny; i++)
    {
        for (int j=0; j<nvx; j++)
        {
            for (int k=0; k<nvy; k++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\n", y_0[i], vx_0[j], vy_0[k], ftle[1][i][j][k]);
            }
        }
    }
}

if (id==21101)
{
    for (int i=0; i<ny; i++)
    {
        for (int j=0; j<nvx; j++)
        {
            for (int k=0; k<ne; k++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\n", y_0[i], vx_0[j], e_0[k], ftle[1][i][j][k]);
            }
        }
    }
}

if (id==21011)
{
    for (int i=0; i<ny; i++)
    {
        for (int j=0; j<nvy; j++)
        {
            for (int k=0; k<ne; k++)
            {
                fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\n", y_0[i], vy_0[j], e_0[k], ftle[1][i][j][k]);
            }
        }
    }
}

 //4D case
if (id==11110)
{
    for (int i=0; i<nx; i++)
    {
        for (int j=0; j<ny; j++)
        {
            for (int k=0; k<nvx; k++)
            {
                for (int l=0; l<nvy; l++)
                {
                    fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\t%.12f\n", x_0[i], y_0[j], vx_0[k], vy_0[l], ftle[i][j][k][l]);
                }
            }
        }
    }
}

if (id==11101)
{
    for (int i=0; i<nx; i++)
    {
        for (int j=0; j<ny; j++)
        {
            for (int k=0; k<nvx; k++)
            {
                for (int l=0; l<ne; l++)
                {
                    fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\t%.12f\n", x_0[i], y_0[j], vx_0[k], e_0[l], ftle[i][j][k][l]);
                }
            }
        }
    }
}

if (id==11011)
{
    for (int i=0; i<nx; i++)
    {
        for (int j=0; j<ny; j++)
        {
            for (int k=0; k<nvy; k++)
            {
                for (int l=0; l<ne; l++)
                {
                    fprintf(ftle_stream,"%.12f\t%.12f\t%.12f\t%.12f\t%.12f\n", x_0[i], y_0[j], vy_0[k], e_0[l], ftle[i][j][k][l]);
                }
            }
        }
    }
}

fclose(ftle_stream);

X_state.clear();
X_state.shrink_to_fit();
return 0;
}
