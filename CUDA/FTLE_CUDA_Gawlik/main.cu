#include "hip/hip_runtime.h"
//General libraries

#include <time.h>
#include <iostream>
#include <math.h>
#include <stdio.h>

//Custom headers
#include "Phase_point.h"
#include "Lagrangian_points.h"

//GSL libraries
#include <gsl/gsl_matrix.h>
#include <gsl/gsl_eigen.h>
#include <gsl/gsl_complex.h>
#include <gsl/gsl_complex_math.h>
#include <gsl/gsl_cblas.h>
#include <gsl/gsl_blas.h>
#include <fstream>

//Boost libraries
#include <boost/numeric/odeint.hpp>

//Thrust libraries
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/remove.h>

#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>

using namespace boost::numeric::odeint;

typedef double value_type;
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;
/*
typedef runge_kutta_fehlberg78< state_type , value_type ,
                                state_type , value_type ,
                                thrust_algebra , thrust_operations > stepper_type;
*/
typedef runge_kutta_dopri5< state_type , value_type ,
                            state_type , value_type ,
                            thrust_algebra , thrust_operations > stepper_type;

const int n=150;
double t0=2*M_PI/3;
double DT=2.0;

// Integrator precision
double abs_tol=1e-6;
double rel_tol=1e-6;
const value_type mu=0.1;
const value_type ecc=0.04;


struct er3bp_system
{

    struct er3bp_functor
    {
        double m_t;
        er3bp_functor( double t ) : m_t(t){}

        template< class T >
        __host__ __device__
        void operator()( T tpl ) const
        {
            value_type x = thrust::get< 0 >( tpl );
            value_type y = thrust::get< 1 >( tpl );
            value_type vx = thrust::get< 2 >( tpl );
            value_type vy = thrust::get< 3 >( tpl );
            thrust::get< 4 >(tpl)=vx; // set tuple's sixth element to vx
            thrust::get< 5 >(tpl)=vy;
            thrust::get< 6 >(tpl)=2*vy+(x-((1-mu)*(x+mu))/pow((x+mu)*(x+mu)+y*y,1.5)-
                                    (mu*(x-1+mu))/pow((x-1+mu)*(x-1+mu)+y*y,1.5))/(1+ecc*cos(m_t));
            thrust::get< 7 >(tpl)=-2*vx+(y-(1-mu)*y/pow((x+mu)*(x+mu)+y*y,1.5)-
                                    mu*y/pow((x-1+mu)*(x-1+mu)+y*y,1.5))/(1+ecc*cos(m_t));
        }
    };


    er3bp_system( size_t N ): m_N( N ) { }

    template< class State , class Deriv >
    void operator()( const State &x , Deriv &dxdt , value_type t ) const
    {
        thrust::for_each(
        // Create each tuple
                thrust::make_zip_iterator( thrust::make_tuple(
                        boost::begin( x ) ,             // x
                        boost::begin( x ) + m_N ,       // y
                        boost::begin( x ) + 2 * m_N ,   // vx
                        boost::begin( x ) + 3 * m_N,    // vy
                        boost::begin( dxdt ) ,
                        boost::begin( dxdt ) + m_N ,
                        boost::begin( dxdt ) + 2 * m_N,
                        boost::begin( dxdt ) + 3 * m_N ) ) ,
                thrust::make_zip_iterator( thrust::make_tuple(
                        boost::begin( x ) + m_N ,
                        boost::begin( x ) + 2 * m_N ,
                        boost::begin( x ) + 3 * m_N ,
                        boost::begin( x ) + 4 * m_N ,
                        boost::begin( dxdt ) + m_N ,
                        boost::begin( dxdt ) + 2 * m_N ,
                        boost::begin( dxdt ) + 3 * m_N ,
                        boost::begin( dxdt ) + 4 * m_N ) ) ,
                er3bp_functor(t) );
    }

size_t m_N;
};


struct Lagrangian_points* Lagr(double mu);
double Omega (double x, double y, double mu);


const int nx=n;
const int ny=3;
const int nvx=n;
const int ne=3;
const int n_tot=nx*ny*nvx*ne;

typedef  bool bool_matrix4d[nx][ny][nvx][ne];
typedef double matrix4d[nx][ny][nvx][ne];

bool_matrix4d filter;
bool_matrix4d filter_ftle;
phase_point pf[nx][ny][nvx][ne];
matrix4d ftle;

int main ()
{
double tempo;

  // Computation parameters
double Tf=t0+DT;
//double ecc=0.0;
double Ki;
Ki=1+ecc*cos(0.0);

  // Initial conditions ranges
  // Assigned initial conditions
double mu=0.10;
struct Lagrangian_points Lagrp;

Lagrp=*Lagr(mu);

double C_L1;
C_L1=2*Omega(Lagrp.xl1,Lagrp.yl1,mu)/Ki; //FIXME devo mettere /Ki?
//std::cout<<C_L1<<'\n';
double E_0=-C_L1/2.0+0.03715;
printf("C_L1 = %.5f E_0 = %.5f Ki = %.5f",C_L1,E_0,Ki);
//std::cout<<E_0; // FIXME generalizzare!

//std::cout<<E_0<<'\n';
double Y_0=0;

//int nx=n;
double x_0_min=-0.8;
double x_0_max=-0.15;
double dx=(x_0_max-x_0_min)/(nx-1);
double x_0[nx];
x_0[0]=x_0_min;
//std::cout<<"x_0[0]="<<x_0[0]<<'\t';
for (int i=1; i<nx ; i++)
{
    x_0[i]=x_0[i-1]+dx;
    //std::cout<<"x_0["<<i<<"]="<<x_0[i]<<'\t';
}

//int nvx=n;
double vx_0_min=-2;
double vx_0_max=2;
double dvx=(vx_0_max-vx_0_min)/(nvx-1);
double vx_0[nvx];
vx_0[0]=vx_0_min;
for (int i=1; i<nvx; i++)
{
    vx_0[i]=vx_0[i-1]+dvx;
}

//int ny=3;
double dy=(dx+dvx)/2; // FIXME assunzione sparata a caso
double y_0[3]={Y_0-dy, Y_0, Y_0+dy}; //FIXME da generalizzare!

//int ne=3;
double de=dy;  //FIXME assunzione
double e_0[3]={E_0-de, E_0, E_0+de}; //FIXME da generalizzare!

double vy_0;

thrust::device_vector< double > X_state(5*n_tot,0); //5: x,y,vx,vy,e
/*
value_type X[n_tot];
value_type Y[n_tot];
value_type VX[n_tot];
value_type VY[n_tot];
value_type E[n_tot];
*/
    //Construct vector for integration
int c=0;
for (int i=0; i<nx; i++)
{
    for (int j=0; j<ny; j++)
    {
        for (int k=0; k<nvx; k++)
        {
            for (int l=0; l<ne; l++)
            {
                vy_0=-sqrt(2*Omega(x_0[i],y_0[j],mu)/Ki+2*e_0[l]-vx_0[k]*vx_0[k]);
                if (!((j!=1) && (l!=1)) && (vy_0<0))
                {
                    X_state[c]=x_0[i];
                    X_state[c+n_tot]=y_0[j];
                    X_state[c+2*n_tot]=vx_0[k];
                    X_state[c+3*n_tot]=vy_0;
                    X_state[c+4*n_tot]=e_0[l];
                    filter[i][j][k][l]=1;
                    filter_ftle[i][j][k][l]=1;
                    c=c+1;
                }
                else
                {
                    filter[i][j][k][l]=0;
                    filter_ftle[i][j][k][l]=0;
                }
            }
        }
    }
}
int dim=c;

for(int i=0;i<dim;i++)
{
    X_state[i+dim]=X_state[i+n_tot];
}
for(int i=0;i<dim;i++)
{
    X_state[i+2*dim]=X_state[i+2*n_tot];
}
for(int i=0;i<dim;i++)
{
    X_state[i+3*dim]=X_state[i+3*n_tot];
}
for(int i=0;i<dim;i++)
{
    X_state[i+4*dim]=X_state[i+4*n_tot];
}

//X_state_host.resize(5*dim);

X_state.resize(5*dim);

/*
for(int i=0; i<dim;i++)
{
    X_state_host[i]=X[i];
    X_state_host[i+dim]=Y[i];
    X_state_host[i+2*dim]=VX[i];
    X_state_host[i+3*dim]=VY[i];
    X_state_host[i+4*dim]=E[i];
}

thrust::device_vector< value_type > X_state=X_state_host;
//thrust::remove(X.begin()+dim+1,X.begin()+n_tot,0);
*/

er3bp_system sistema (dim);
clock_t start=clock();
double steps;
steps = integrate_adaptive(
        make_controlled(abs_tol , rel_tol, stepper_type() ),
        sistema ,
        X_state ,
        t0 ,
        Tf ,
        1e-6);

std::cout<<steps<<'\n';
clock_t end=clock();
tempo=((double)(end-start))/CLOCKS_PER_SEC;
std::cout<<"integration time "<<tempo<<'\n';

    //Deconstruct integration results
    start=clock();

c=0;
for (int i=0; i<nx; i++)
{
    for (int j=0; j<ny; j++)
    {
        for (int k=0; k<nvx; k++)
        {
            for (int l=0; l<ne; l++)
            {
                pf[i][j][k][l].x=0;
                pf[i][j][k][l].y=0;
                pf[i][j][k][l].vx=0;
                pf[i][j][k][l].vy=0;
                pf[i][j][k][l].e=0;
                if (filter[i][j][k][l])
                {
                    pf[i][j][k][l].x=X_state[c];
                    pf[i][j][k][l].y=X_state[c+dim];
                    pf[i][j][k][l].vx=X_state[c+2*dim];
                    pf[i][j][k][l].vy=X_state[c+3*dim];
                    pf[i][j][k][l].e=0.5*((X_state[c+2*dim]*X_state[c+2*dim])+
                                        (X_state[c+3*dim]*X_state[c+3*dim]))-
                                        Omega(X_state[c],X_state[c+dim],mu)/(1+ecc*cos(Tf));
                    c=c+1;
                }
                if (i==0 || i==nx || j==0 || j==ny || k==0 || k==nvx || l==0 || l==ne)
                    {
                    filter_ftle[i][j][k][l]=0;
                    }
                else{
                    if (filter[i][j][k][l]==0)
                    {
                    filter_ftle[i][j][k][l]=0;
                    filter_ftle[i+1][j][k][l]=0;
                    filter_ftle[i-1][j][k][l]=0;
                    filter_ftle[i][j+1][k][l]=0;
                    filter_ftle[i][j-1][k][l]=0;
                    filter_ftle[i][j][k+1][l]=0;
                    filter_ftle[i][j][k-1][l]=0;
                    filter_ftle[i][j][k][l+1]=0;
                    filter_ftle[i][j][k][l-1]=0;
                    }
                }
            }
        }
    }
}
end=clock();
tempo=((double)(end-start))/CLOCKS_PER_SEC;
std::cout<<"Deconstruction time "<<tempo<<'\n';
    //FTLE computation
//matrix4d ftle;
gsl_matrix *dphi=gsl_matrix_alloc (4, 4);
gsl_matrix *C=gsl_matrix_alloc (4,4);
double l_max;
//double alpha=1;
gsl_vector *eval = gsl_vector_alloc (4);
//gsl_matrix *evec = gsl_matrix_alloc (4, 4);
gsl_eigen_symm_workspace * w = gsl_eigen_symm_alloc (4);
for (int i=0; i<nx; i++)
{
    for (int j=0; j<ny; j++)
    {
        for (int k=0; k<nvx; k++)
        {
            for (int l=0; l<ne; l++)
            {
                ftle[i][j][k][l]=0;
                if (filter_ftle[i][j][k][l])
                {
                gsl_matrix_set(dphi,0,0,(pf[i+1][j][k][l].x-pf[i-1][j][k][l].x)/(2*dx));
                gsl_matrix_set(dphi,0,1,(pf[i][j+1][k][l].x-pf[i][j-1][k][l].x)/(2*dy));
                gsl_matrix_set(dphi,0,2,(pf[i][j][k+1][l].x-pf[i][j][k-1][l].x)/(2*dvx));
                gsl_matrix_set(dphi,0,3,(pf[i][j][k][l+1].x-pf[i][j][k][l-1].x)/(2*de));

                gsl_matrix_set(dphi,1,0,(pf[i+1][j][k][l].y-pf[i-1][j][k][l].y)/(2*dx));
                gsl_matrix_set(dphi,1,1,(pf[i][j+1][k][l].y-pf[i][j-1][k][l].y)/(2*dy));
                gsl_matrix_set(dphi,1,2,(pf[i][j][k+1][l].y-pf[i][j][k-1][l].y)/(2*dvx));
                gsl_matrix_set(dphi,1,3,(pf[i][j][k][l+1].y-pf[i][j][k][l-1].y)/(2*de));

                gsl_matrix_set(dphi,2,0,(pf[i+1][j][k][l].vx-pf[i-1][j][k][l].vx)/(2*dx));
                gsl_matrix_set(dphi,2,1,(pf[i][j+1][k][l].vx-pf[i][j-1][k][l].vx)/(2*dy));
                gsl_matrix_set(dphi,2,2,(pf[i][j][k+1][l].vx-pf[i][j][k-1][l].vx)/(2*dvx));
                gsl_matrix_set(dphi,2,3,(pf[i][j][k][l+1].vx-pf[i][j][k][l-1].vx)/(2*de));

                gsl_matrix_set(dphi,3,0,(pf[i+1][j][k][l].e-pf[i-1][j][k][l].e)/(2*dx));
                gsl_matrix_set(dphi,3,1,(pf[i][j+1][k][l].e-pf[i][j-1][k][l].e)/(2*dy));
                gsl_matrix_set(dphi,3,2,(pf[i][j][k+1][l].e-pf[i][j][k-1][l].e)/(2*dvx));
                gsl_matrix_set(dphi,3,3,(pf[i][j][k][l+1].e-pf[i][j][k][l-1].e)/(2*de));

                gsl_blas_dgemm(CblasTrans,CblasNoTrans,1.0,dphi,dphi,0,C);

                gsl_eigen_symm (C, eval, w);

                gsl_vector_set(eval,0,fabs(gsl_vector_get(eval,0)));
                gsl_vector_set(eval,1,fabs(gsl_vector_get(eval,1)));
                gsl_vector_set(eval,2,fabs(gsl_vector_get(eval,2)));
                gsl_vector_set(eval,3,fabs(gsl_vector_get(eval,3)));
                l_max=gsl_vector_max(eval);
                ftle[i][j][k][l]=1/fabs(DT)*log(sqrt(l_max));
                }
            }
        }
    }
}
FILE *ftle_stream;
char s[100];
sprintf(s,"ftle_ell_n=%.0i_t=%.2f.txt",n,t0);
ftle_stream=fopen(s,"w");
for (int i=0; i<nx; i++)
{   fprintf(ftle_stream,"\n");
    for (int k=0; k<nvx; k++)
    {
        fprintf(ftle_stream,"%.12f\t",ftle[i][1][k][1]);
    }
}

fclose(ftle_stream);
return 0;
}
